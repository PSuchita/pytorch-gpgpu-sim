
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathMagma.cu"
#else

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)

#ifdef USE_MAGMA

static void THCTensor_(copyArray1d)(THCState *state, THCTensor *self, real *src, int k)
{
  int64_t size[1] = { k };
  int64_t stride[1] = { 1 };
  THCTensor_(resizeNd)(state, self, 1, size, stride);
  size_t len = k * sizeof(real);
  THCudaCheck(cudaMemcpy(THCStorage_(data)(state, self->storage) + self->storageOffset, src, len, cudaMemcpyHostToDevice));
}

static void THCTensor_(copyArray2d)(THCState *state, THCTensor *self, real *src, int m, int n)
{
  int64_t size[2] = { m, n };
  int64_t stride[2] = { 1, m };
  THCTensor_(resizeNd)(state, self, 2, size, stride);
  size_t len = m * n * sizeof(real);
  THCudaCheck(cudaMemcpy(THCStorage_(data)(state, self->storage) + self->storageOffset, src, len, cudaMemcpyHostToDevice));
}

static void THCTensor_(copyTensor2d)(THCState *state, real *dst, THCTensor *self)
{
  THAssert(self->_dim() == 2);
  size_t len = THCTensor_(nElement)(state, self)*sizeof(real);
  THCTensor *temp = THCTensor_(newTranspose)(state, self, 0, 1);
  THCTensor *selfc = THCTensor_(newContiguous)(state, temp);
  THCudaCheck(cudaMemcpy(dst, THCStorage_(data)(state, selfc->storage) + selfc->storageOffset, len, cudaMemcpyDeviceToHost));
  THCTensor_(free)(state, temp);
  THCTensor_(free)(state, selfc);
}

#endif // USE_MAGMA

static THCTensor* THCTensor_(newColumnMajor)(THCState *state, THCTensor *self, THCTensor *src)
{
  THAssert(src->_dim() == 2);
  if (self == src && self->stride[0] == 1 && self->stride[1] == self->size[0])
  {
    THCTensor_(retain)(state, self);
    return self;
  }

  if (self == src)
    self = THCTensor_(new)(state);
  else
    THCTensor_(retain)(state, self);

  int64_t size[2] = { src->size[0], src->size[1] };
  int64_t stride[2] = { 1, src->size[0] };

  THCTensor_(resizeNd)(state, self, 2, size, stride);
  THCTensor_(copy)(state, self, src);
  return self;
}


THC_API void THCTensor_(gesv)(THCState *state, THCTensor *rb_, THCTensor *ra_, THCTensor *b_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 1, "A should be (non-empty) 2 dimensional");
  THArgCheck(!b_->is_empty() && b_->dim() == 2, 2, "b should be (non-empty) 2 dimensional");
  THArgCheck(a_->size[0] == a_->size[1], 1, "A should be square");
  THArgCheck(b_->size[0] == a_->size[0], 2, "A,b size incompatible");

  int64_t n = a_->size[0];
  int64_t nrhs = b_->size[1];

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  THCTensor *b = THCTensor_(newColumnMajor)(state, rb_, b_);
  real *a_data = THCTensor_(data)(state, a);
  real *b_data = THCTensor_(data)(state, b);

  int *ipiv = th_magma_malloc_pinned<int>(n);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgesv_gpu(n, nrhs, a_data, n, ipiv, b_data, n, &info);
#else
  magma_dgesv_gpu(n, nrhs, a_data, n, ipiv, b_data, n, &info);
#endif

  if (info < 0)
    THError("MAGMA gesv : Argument %d : illegal value", -info);
  else if (info > 0)
    THError("MAGMA gesv : U(%d,%d) is zero, singular U.", info, info);

  magma_free_pinned(ipiv);
  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(freeCopyTo)(state, b, rb_);
#else
  THError(NoMagma(gesv));
#endif
}

THC_API void THCTensor_(trtrs)(THCState *state, THCTensor *rb_, THCTensor *ra_, THCTensor *b_, THCTensor *a_,
                               const char *uplo, const char *trans, const char *diag)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 1, "A should be (non-empty) 2 dimensional");
  THArgCheck(!b_->is_empty() && b_->dim() == 2, 2, "b should be (non-empty) 2 dimensional");
  THArgCheck(a_->size[0] == a_->size[1], 1, "A should be square");
  THArgCheck(b_->size[0] == a_->size[0], 2, "A,b size incompatible");

  magma_side_t sz = MagmaLeft;
  magma_uplo_t ul = uplo[0] == 'U' ?  MagmaUpper : MagmaLower;
  magma_trans_t ts = trans[0] == 'N' ? MagmaNoTrans : MagmaTrans;
  magma_diag_t dg = diag[0] == 'U' ? MagmaUnit : MagmaNonUnit;

  real alpha = 1;

  int64_t n = a_->size[0];
  int64_t nrhs = b_->size[1];

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  THCTensor *b = THCTensor_(newColumnMajor)(state, rb_, b_);
  real *a_data = THCTensor_(data)(state, a);
  real *b_data = THCTensor_(data)(state, b);

#if defined(THC_REAL_IS_FLOAT)
  magma_strsm(sz, ul, ts, dg, n, nrhs, alpha, a_data, n, b_data, n);
#else
  magma_dtrsm(sz, ul, ts, dg, n, nrhs, alpha, a_data, n, b_data, n);
#endif

  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(freeCopyTo)(state, b, rb_);
#else
  THError(NoMagma(trtrs));
#endif
}

THC_API void THCTensor_(gels)(THCState *state, THCTensor *rb_, THCTensor *ra_, THCTensor *b_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 1, "A should be (non-empty) 2 dimensional");
  THArgCheck(!b_->is_empty() && b_->dim() == 2, 1, "b should be (non-empty) 2 dimensional");
  THArgCheck(a_->size[0] == b_->size[0], 2, "Expected A and b to have same size "
      "at dim 0, but they have incompatible sizes");
  THArgCheck(a_->size[0] >= a_->size[1], 2, "Expected A with shape (m x n) to have "
      "m >= n. The case for m < n is not implemented yet.");

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  THCTensor *b = THCTensor_(newColumnMajor)(state, rb_, b_);
  real *a_data = THCTensor_(data)(state, a);
  real *b_data = THCTensor_(data)(state, b);

  int64_t m = a->size[0];
  int64_t n = a->size[1];
  int64_t nrhs = b->size[1];
  real wkopt;

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);
#else
  magma_dgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);
#endif

  real *hwork = th_magma_malloc_pinned<real>((size_t)wkopt);

#if defined(THC_REAL_IS_FLOAT)
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
#else
  magma_dgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
#endif

  magma_free_pinned(hwork);

  if (info != 0)
    THError("MAGMA gels : Argument %d : illegal value", -info);

  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(freeCopyTo)(state, b, rb_);
#else
  THError(NoMagma(gels));
#endif
}

THC_API void THCTensor_(syev)(THCState *state, THCTensor *re_, THCTensor *rv_, THCTensor *a, const char *jobzs, const char *uplos)
{
#ifdef USE_MAGMA
  int64_t n = a->size[0];
  int64_t lda = n;

  magma_uplo_t uplo = uplos[0] == 'U' ?  MagmaUpper : MagmaLower;
  magma_vec_t jobz = jobzs[0] == 'N' ? MagmaNoVec : MagmaVec;

  THCTensor *input = THCTensor_(newColumnMajor)(state, rv_, a);
  real *input_data = THCTensor_(data)(state, input);

  // eigen values and workspace
  real *w = th_magma_malloc_pinned<real>(n);
  real *wA = th_magma_malloc_pinned<real>(lda * n);

  // compute optimal size of work array
  int info;
  real lwork;
  int liwork;

#if defined(THC_REAL_IS_FLOAT)
  magma_ssyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, &lwork, -1, &liwork, -1, &info);
#else
  magma_dsyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, &lwork, -1, &liwork, -1, &info);
#endif

  real *work = th_magma_malloc_pinned<real>((size_t)lwork);
  int *iwork = th_magma_malloc_pinned<int>(liwork);

  // compute eigenvalues and, optionally, eigenvectors
#if defined(THC_REAL_IS_FLOAT)
  magma_ssyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, work, (int) lwork, iwork, liwork, &info);
#else
  magma_dsyevd_gpu(jobz, uplo, n, input_data, lda, w, wA, n, work, (int) lwork, iwork, liwork, &info);
#endif

  // copy eigen values from w to re_
  if (info == 0)
    THCTensor_(copyArray1d)(state, re_, w, n);

  magma_free_pinned(iwork);
  magma_free_pinned(work);
  magma_free_pinned(wA);
  magma_free_pinned(w);

  // check error value
  if (info > 0)
    THError("MAGMA syev : Failed to converge. %d off-diagonal elements of an didn't converge to zero", info);
  else if (info < 0)
    THError("MAGMA syev : Argument %d : illegal value", -info);

  THCTensor_(freeCopyTo)(state, input, rv_);
#else
  THError(NoMagma(syev));
#endif
}

THC_API void THCTensor_(geev)(THCState *state, THCTensor *re_, THCTensor *rv_, THCTensor *a_, const char *jobvrs)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 3, "A should be (non-empty) 2 dimensional");
  THArgCheck(a_->size[0] == a_->size[1], 3, "A should be square");

  magma_vec_t jobvr = jobvrs[0] == 'N' ? MagmaNoVec : MagmaVec;
  int64_t n = a_->size[0];

  real *a_data = th_magma_malloc_pinned<real>(n * n);
  THCTensor_(copyTensor2d)(state, a_data, a_);

  real *wr = th_magma_malloc_pinned<real>(n);
  real *wi = th_magma_malloc_pinned<real>(n);

  real *vr_data = NULL;
  int64_t ldvr = 1;
  if (jobvr == MagmaVec)
  {
    vr_data = th_magma_malloc_pinned<real>(n * n);
    ldvr = n;
  }

  real wkopt;
  int info;

#if defined(THC_REAL_IS_FLOAT)
  magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, &wkopt, -1, &info);
#else
  magma_dgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, &wkopt, -1, &info);
#endif

  int lwork = (int) wkopt;
  real *work_data = th_magma_malloc_pinned<real>(lwork);

#if defined(THC_REAL_IS_FLOAT)
  magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, work_data, lwork, &info);
#else
  magma_dgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, work_data, lwork, &info);
#endif

  if (info > 0)
    THError("MAGMA geev : Failed to converge. %d off-diagonal elements of an didn't converge to zero", info);
  else if (info < 0)
    THError("MAGMA geev : Argument %d : illegal value", -info);

  {
    THCTensor_(resize2d)(state, re_, 2, n);
    THCTensor *re = THCTensor_(newContiguous)(state, re_);
    THCudaCheck(cudaMemcpy(THCStorage_(data)(state, re->storage) + re->storageOffset, wr, n*sizeof(real), cudaMemcpyHostToDevice));
    THCudaCheck(cudaMemcpy(THCStorage_(data)(state, re->storage) + re->storageOffset + n, wi, n*sizeof(real), cudaMemcpyHostToDevice));
    THCTensor_(freeCopyTo)(state, re, re_);
    THCTensor_(transpose)(state, re_, NULL, 0, 1);
  }

  if (jobvr == MagmaVec)
    THCTensor_(copyArray2d)(state, rv_, vr_data, n, n);

  magma_free_pinned(work_data);
  magma_free_pinned(vr_data);
  magma_free_pinned(wi);
  magma_free_pinned(wr);
  magma_free_pinned(a_data);

#else
  THError(NoMagma(geev));
#endif
}

THC_API void THCTensor_(gesvd)(THCState *state, THCTensor *ru_, THCTensor *rs_, THCTensor *rv_, THCTensor *a, const char *jobu)
{
#ifdef USE_MAGMA
  THCTensor *ra_ = THCTensor_(new)(state);
  THCTensor_(gesvd2)(state, ru_, rs_, rv_,  ra_, a, jobu);
  THCTensor_(free)(state, ra_);
#else
  THError(NoMagma(gesvd));
#endif
}

THC_API void THCTensor_(gesvd2)(THCState *state, THCTensor *ru_, THCTensor *rs_, THCTensor *rv_, THCTensor *ra_, THCTensor *a, const char *jobus)
{
#ifdef USE_MAGMA
  THArgCheck(!a->is_empty() && a->dim() == 2, 2, "A should be non-empty 2 dimensional");

  magma_vec_t jobz = jobus[0] == 'A' ? MagmaAllVec : jobus[0] == 'S' ? MagmaSomeVec : jobus[0] == 'O' ? MagmaOverwriteVec : MagmaNoVec;

  int iunused[1];
  int64_t m = a->size[0];
  int64_t n = a->size[1];
  int64_t k = m < n ? m : n;
  int64_t j = (jobz == MagmaAllVec) ? m : k;
  int64_t jv = (jobz == MagmaAllVec) ? n : k;

  real *a_data = th_magma_malloc_pinned<real>(m * n);
  THCTensor_(copyTensor2d)(state, a_data, a);

  real *rs_data = th_magma_malloc_pinned<real>(k);
  real *ru_data = th_magma_malloc_pinned<real>(m * j);
  real *rv_data = th_magma_malloc_pinned<real>(n * n);

  real wkopt;
  int info;

#if defined(THC_REAL_IS_FLOAT)
  magma_sgesdd(jobz, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, &wkopt, -1, iunused, &info);
#else
  magma_dgesdd(jobz, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, &wkopt, -1, iunused, &info);
#endif

  int lwork = (int) wkopt;
  real *work_data = th_magma_malloc_pinned<real>(lwork);
  int *iwork = th_magma_malloc_pinned<int>(8 * k);

#if defined(THC_REAL_IS_FLOAT)
  magma_sgesdd(jobz, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, work_data, lwork, iwork, &info);
#else
  magma_dgesdd(jobz, m, n, a_data, m, rs_data, ru_data, m, rv_data, n, work_data, lwork, iwork, &info);
#endif

  if (info > 0)
    THError("MAGMA gesdd : the updating process of SBDSDC did not converge (error: %d)", info);
  else if (info < 0)
    THError("MAGMA gesdd : Argument %d : illegal value", -info);

  THCTensor_(copyArray2d)(state, rv_, rv_data, n, n);
  THCTensor_(transpose)(state, rv_, NULL, 0, 1);
  if (jobz != MagmaAllVec)
    THCTensor_(narrow)(state, rv_, rv_, 1, 0, jv);
  THCTensor_(copyArray2d)(state, ru_, ru_data, m, j);
  THCTensor_(copyArray1d)(state, rs_, rs_data, k);
  THCTensor_(copyArray2d)(state, ra_, a_data,  m, n);

  magma_free_pinned(work_data);
  magma_free_pinned(iwork);
  magma_free_pinned(rv_data);
  magma_free_pinned(ru_data);
  magma_free_pinned(rs_data);
  magma_free_pinned(a_data);
#else
  THError(NoMagma(gesvd2));
#endif
}

THC_API void THCTensor_(getri)(THCState *state, THCTensor *ra_, THCTensor *a)
{
  THArgCheck(!a->is_empty() && a->dim() == 2, 2, "A should be non-empty 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

#ifdef USE_MAGMA
  int info;
  int64_t n = a->size[0];
  int lwork = n * magma_get_sgetri_nb(n);

  THCTensor *input = THCTensor_(newColumnMajor)(state, ra_, a);
  real *input_data = THCTensor_(data)(state, input);

  int *ipiv = th_magma_malloc_pinned<int>(n);

  THCTensor *work = THCTensor_(newWithSize1d)(state, lwork);
  real *work_data = THCTensor_(data)(state, work);

  // Run LU
#if defined(THC_REAL_IS_FLOAT)
  magma_sgetrf_gpu(n, n, input_data, n, ipiv, &info);
#else
  magma_dgetrf_gpu(n, n, input_data, n, ipiv, &info);
#endif

  if (info > 0)
    THError("MAGMA getrf : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("MAGMA getrf : Argument %d : illegal value", -info);

  // Inverse
#if defined(THC_REAL_IS_FLOAT)
  magma_sgetri_gpu(n, input_data, n, ipiv, work_data, lwork, &info);
#else
  magma_dgetri_gpu(n, input_data, n, ipiv, work_data, lwork, &info);
#endif

  if (info > 0)
    THError("MAGMA getri : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("MAGMA getri : Argument %d : illegal value", -info);

  THCTensor_(free)(state, work);
  magma_free_pinned(ipiv);
  THCTensor_(freeCopyTo)(state, input, ra_);
#else
  int64_t n = a->size[0];

  // input
  THCTensor *input = THCTensor_(newColumnMajor)(state, a, a);
  THCTensor_(resizeNd)(state, ra_, 2, input->size, input->stride);

  real *matrices1[1] = { THCTensor_(data)(state, input) };
  real *matrices2[1] = { THCTensor_(data)(state, ra_) };

  // Copy pointers to device.
  auto d_matrices1 = static_cast<real**>(THCudaMalloc(state, sizeof(real*)));
  auto d_matrices2 = static_cast<real**>(THCudaMalloc(state, sizeof(real*)));

  THCudaCheck(cudaMemcpyAsync(d_matrices1, matrices1, sizeof(real*),
                              cudaMemcpyHostToDevice, THCState_getCurrentStream(state)));
  THCudaCheck(cudaMemcpyAsync(d_matrices2, matrices2, sizeof(real*),
                              cudaMemcpyHostToDevice, THCState_getCurrentStream(state)));
  int info;
  auto info_gpu = static_cast<int*>(THCudaMalloc(state, sizeof(int)));

  auto ipiv_gpu = static_cast<int*>(THCudaMalloc(state, n * sizeof(int)));

  // Run LU
#if defined(THC_REAL_IS_FLOAT)
  THCudaBlas_Sgetrf(state, n, d_matrices1, n, ipiv_gpu, info_gpu, 1);
#else
  THCudaBlas_Dgetrf(state, n, d_matrices1, n, ipiv_gpu, info_gpu, 1);
#endif

  THCudaCheck(cudaMemcpy(&info, info_gpu, sizeof(int), cudaMemcpyDeviceToHost));

  if (info > 0)
    THError("CUBLAS getrf : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("CUBLAS getrf : Argument %d : illegal value", -info);

  // Inverse
#if defined(THC_REAL_IS_FLOAT)
  THCudaBlas_Sgetri(state, n, (const real**)d_matrices1, n, ipiv_gpu, d_matrices2, n, info_gpu, 1);
#else
  THCudaBlas_Dgetri(state, n, (const real**)d_matrices1, n, ipiv_gpu, d_matrices2, n, info_gpu, 1);
#endif

  THCudaCheck(cudaMemcpy(&info, info_gpu, sizeof(int), cudaMemcpyDeviceToHost));

  if (info > 0)
    THError("CUBLAS getri : U(%d,%d) is 0, U is singular", info, info);
  else if (info < 0)
    THError("CUBLAS getri : Argument %d : illegal value", -info);

  THCudaFree(state, ipiv_gpu);
  THCudaFree(state, info_gpu);

  THCudaFree(state, d_matrices1);
  THCudaFree(state, d_matrices2);

  THCTensor_(free)(state, input);
#endif
}

__global__ void THCTensor_(copyUpperSymmetric)(real *input, int n, int len)
{
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += 65535) {
    const int r = idx % n;
    const int c = idx / n;
    if (r > c) {
      input[idx] = input[r*n + c];
    }
  }
}

__global__ void THCTensor_(copyLowerSymmetric)(real *input, int n, int len)
{
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += 65535) {
    const int r = idx % n;
    const int c = idx / n;
    if (r < c) {
      input[idx] = input[r*n + c];
    }
  }
}

THC_API void THCTensor_(potri)(THCState *state, THCTensor *ra_, THCTensor *a, const char *uplo)
{
#ifdef USE_MAGMA
  THArgCheck(!a->is_empty() && a->dim() == 2, 2, "A should be non-empty 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int64_t n = a->size[0];
  magma_uplo_t ul = uplo[0] == 'U' ?  MagmaUpper : MagmaLower;

  THCTensor *input = THCTensor_(newColumnMajor)(state, ra_, a);
  real *input_data = THCTensor_(data)(state, input);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_spotri_gpu(ul, n, input_data, n, &info);
#else
  magma_dpotri_gpu(ul, n, input_data, n, &info);
#endif

  if (info > 0)
    THError("MAGMA potri : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potri : Argument %d : illegal value", -info);

  cudaStream_t stream = THCState_getCurrentStream(state);
  const int len = n*n;
  dim3 blocks(std::min(DIVUP(len, 128), 65535));
  dim3 threads(128);
  if (uplo[0] == 'U') {
    THCTensor_(copyUpperSymmetric)<<<blocks, threads, 0, stream>>>(input_data, n, len);
  } else {
    THCTensor_(copyLowerSymmetric)<<<blocks, threads, 0, stream>>>(input_data, n, len);
  }

  THCTensor_(freeCopyTo)(state, input, ra_);
#else
  THError(NoMagma(potri));
#endif
}

THC_API void THCTensor_(potrf)(THCState *state, THCTensor *ra_, THCTensor *a, const char *uplo)
{
#ifdef USE_MAGMA
  THArgCheck(!a->is_empty() && a->dim() == 2, 2, "A should be (non-empty) 2 dimensional");
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int64_t n = a->size[0];
  magma_uplo_t ul = uplo[0] == 'U' ?  MagmaUpper : MagmaLower;

  THCTensor *input = THCTensor_(newColumnMajor)(state, ra_, a);
  real *input_data = THCTensor_(data)(state, input);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_spotrf_gpu(ul, n, input_data, n, &info);
#else
  magma_dpotrf_gpu(ul, n, input_data, n, &info);
#endif

  // check error value
  if (info > 0)
    THError("MAGMA potrf : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potrf : Argument %d : illegal value", -info);

  if (uplo[0] == 'U') {
    THCTensor_(triu)(state, ra_, input, 0);
  } else {
    THCTensor_(tril)(state, ra_, input, 0);
  }
  THCTensor_(free)(state, input);
#else
  THError(NoMagma(potrf));
#endif
}

THC_API void THCTensor_(potrs)(THCState *state, THCTensor *rb_, THCTensor *b, THCTensor *a, const char *uplo)
{
#ifdef USE_MAGMA
  THArgCheck(a->size[0] == a->size[1], 2, "A should be square");

  int64_t n = a->size[0];
  int64_t nrhs = b->size[1];
  magma_uplo_t ul = uplo[0] == 'U' ?  MagmaUpper : MagmaLower;

  THCTensor *b_ = THCTensor_(newColumnMajor)(state, rb_, b);
  real *b_data = THCTensor_(data)(state, b_);
  THCTensor *a_ = THCTensor_(newColumnMajor)(state, a, a);
  real *a_data = THCTensor_(data)(state, a_);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_spotrs_gpu(ul, n, nrhs, a_data, n, b_data, n, &info);
#else
  magma_dpotrs_gpu(ul, n, nrhs, a_data, n, b_data, n, &info);
#endif

  // check error value
  if (info < 0)
    THError("MAGMA potrs : Argument %d : illegal value", -info);

  THCTensor_(freeCopyTo)(state, b_, rb_);
  THCTensor_(free)(state, a_);
#else
  THError(NoMagma(potrs));
#endif
}

THC_API void THCTensor_(geqrf)(THCState *state, THCTensor *ra_, THCTensor *rtau_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 2, "A should be non-empty 2 dimensional");

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  int64_t m = a->size[0];
  int64_t n = a->size[1];
  int64_t k = (m < n ? m : n);

#if defined(THC_REAL_IS_FLOAT)
  int64_t nb = magma_get_sgeqrf_nb(m, n);
#else
  int64_t nb = magma_get_dgeqrf_nb(m, n);
#endif

  real *rtau_data = th_magma_malloc_pinned<real>(k);
  real *a_data = THCTensor_(data)(state, a);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgeqrf2_gpu(m, n, a_data, m, rtau_data, &info);
#else
  magma_dgeqrf2_gpu(m, n, a_data, m, rtau_data, &info);
#endif

  if (info != 0)
    THError("MAGMA geqrf2 : Argument %d : illegal value.", -info);

  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(copyArray1d)(state, rtau_, rtau_data, k);
  magma_free_pinned(rtau_data);
#else
  THError(NoMagma(geqrf));
#endif
}

THC_API void THCTensor_(qr)(THCState *state, THCTensor *rq_, THCTensor *rr_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 2, "A should be non-empty 2 dimensional");

  THCTensor *a = THCTensor_(newColumnMajor)(state, rr_, a_);
  int64_t m = a->size[0];
  int64_t n = a->size[1];
  int64_t k = (m < n ? m : n);

#if defined(THC_REAL_IS_FLOAT)
  int64_t nb = magma_get_sgeqrf_nb(m, n);
#else
  int64_t nb = magma_get_dgeqrf_nb(m, n);
#endif

  real *a_data = THCTensor_(data)(state, a);
  real *tau_data = th_magma_malloc_pinned<real>(k);
  THCTensor *work = THCTensor_(newWithSize1d)(state, (2*k + magma_roundup(n, 32))*nb);
  real *work_data = THCTensor_(data)(state, work);

  int info;
  // We need to call two different versions of ?geqrf:
  //   ?geqrf_gpu allows fast computation of Q via ?orqrf_gpu, but doesn't give
  //     R properly. Note that the MAGMA documentation for this method is wrong.
  //     http://icl.cs.utk.edu/magma/forum/viewtopic.php?f=2&t=1015&p=2800&hilit=geqrf_gpu#p2800
  //   ?geqrf2_gpu gives correct R, but doesn't allow computation of Q via ?orqrf_gpu
#if defined(THC_REAL_IS_FLOAT)
  magma_sgeqrf2_gpu(m, n, a_data, m, tau_data, &info);
#else
  magma_dgeqrf2_gpu(m, n, a_data, m, tau_data, &info);
#endif

  if (info != 0)
    THError("MAGMA geqrf2 : Argument %d : illegal value.", -info);

  THCTensor_(narrow)(state, a, a, 0, 0, k);
  THCTensor_(triu)(state, rr_, a, 0);
  THCTensor_(free)(state, a);

  a = THCTensor_(newColumnMajor)(state, rq_, a_);
  a_data = THCTensor_(data)(state, a);

#if defined(THC_REAL_IS_FLOAT)
  magma_sgeqrf_gpu(m, n, a_data, m, tau_data, work_data, &info);
#else
  magma_dgeqrf_gpu(m, n, a_data, m, tau_data, work_data, &info);
#endif

  if (info != 0)
    THError("MAGMA geqrf : Argument %d : illegal value.", -info);

  THCTensor *q = THCTensor_(newColumnMajor)(state, rq_, a);
  real *q_data = THCTensor_(data)(state, q);

#if defined(THC_REAL_IS_FLOAT)
  magma_sorgqr_gpu(m, k, k, q_data, m, tau_data, work_data, nb, &info);
#else
  magma_dorgqr_gpu(m, k, k, q_data, m, tau_data, work_data, nb, &info);
#endif

  if (info != 0)
    THError("MAGMA orgqr : Argument %d : illegal value.", -info);

  THCTensor_(free)(state, a);
  THCTensor_(free)(state, work);
  magma_free_pinned(tau_data);

  THCTensor_(narrow)(state, q, q, 1, 0, k);
  THCTensor_(freeCopyTo)(state, q, rq_);
#else
  THError(NoMagma(qr));
#endif
}

#endif

#endif
